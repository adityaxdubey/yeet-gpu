#include <hip/hip_runtime.h>

__global__ void matrix_multiplication_kernel(const float* A, const float* B, float* C, int M, int N, int K) {
    int r=blockIdx.x*blockDim.x+threadIdx.x;
    int c=blockIdx.y*blockDim.y+threadIdx.y;
    if (c<K && r<M){
        float sum=0.0;
        for(int i=0;i<N;i++){
            sum+=A[r*N+i]*B[i*K+c];
        }
        C[r*K+c]=sum;
    }
}

extern "C" void solve(const float* A, const float* B, float* C, int M, int N, int K) {
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((K + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (M + threadsPerBlock.y - 1) / threadsPerBlock.y);
    
    matrix_multiplication_kernel<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, M, N, K);
    hipDeviceSynchronize();
}
