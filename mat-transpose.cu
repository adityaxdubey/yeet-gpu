#include <hip/hip_runtime.h>
__global__ void matrix_transpose_kernel(const float* input, float* output, int rows, int cols){
    int i=blockIdx.x*blockDim.x+threadIdx.x;
    int j=blockIdx.y*blockDim.y+threadIdx.y;
    if(i<cols && j<rows){
        output[i*rows+j]=input[j*cols+i];
    }
}


extern "C" void solve(const float* input,float* output, int rows, int cols) {
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((cols+threadsPerBlock.x-1)/threadsPerBlock.x,
                       (rows+threadsPerBlock.y-1)/threadsPerBlock.y);

    matrix_transpose_kernel<<<blocksPerGrid,threadsPerBlock>>>(input, output, rows, cols);
    hipDeviceSynchronize();
}