#include <hip/hip_runtime.h>

__global__ void copy_matrix_kernel(const float* A, float* B, int N) {
    int y=blockDim.x*blockIdx.x+threadIdx.x;
    if (y<N*N){
        B[y]=A[y];
    }
}

//a and b are device pointers
extern "C" void solve(const float* A, float* B, int N) {
    int total=N*N;
    int threadsPerBlock=256;
    int blocksPerGrid=(total+threadsPerBlock-1)/threadsPerBlock;
    copy_matrix_kernel<<<blocksPerGrid,threadsPerBlock>>>(A,B,N);
    hipDeviceSynchronize();
} 