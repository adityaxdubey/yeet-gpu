#include <hip/hip_runtime.h>

__global__ void reverse_array(float* input, int N) {
    int i=blockIdx.x*blockDim.x+threadIdx.x;
    int j=N-i-1;
    if (i<j){
        float tmp=input[i];
        input[i]=input[j];
        input[j]=tmp; }
}

extern "C" void solve(float* input, int N) {
    int threadsPerBlock=256;
    int blocksPerGrid=(N+threadsPerBlock-1)/threadsPerBlock;

    reverse_array<<<blocksPerGrid,threadsPerBlock>>>(input, N);
    hipDeviceSynchronize();
}