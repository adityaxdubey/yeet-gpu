#include <hip/hip_runtime.h>

__global__ void relu_kernel(const float* input, float* output,int N) {
    int i=blockIdx.x*blockDim.x+threadIdx.x;
    if (i<N){
        output[i]=max(input[i],(float)0.0);}
}


extern "C" void solve(const float* input, float* output,int N) {
    int threadsPerBlock=256;
    int blocksPerGrid=(N+threadsPerBlock-1)/threadsPerBlock;

    relu_kernel<<<blocksPerGrid,threadsPerBlock>>>(input,output,N);
    hipDeviceSynchronize();
}
