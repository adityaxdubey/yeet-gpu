#include <hip/hip_runtime.h>

__global__ void count_equal_kernel(const int* input, int* output, int N, int K) {
    int ind=threadIdx.x+blockIdx.x*blockDim.x;
    if (ind<N && input[ind]==K){
        atomicAdd(output,1);
    }
}

extern "C" void solve(const int* input, int* output, int N, int K) {
    int threadsPerBlock=256;
    int blocksPerGrid=(N+threadsPerBlock-1)/threadsPerBlock;

    count_equal_kernel<<<blocksPerGrid,threadsPerBlock>>>(input,output, N, K);
    hipDeviceSynchronize();
}